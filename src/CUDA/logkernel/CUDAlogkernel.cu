#include "hip/hip_runtime.h"
#include <thrust/complex.h>

using namespace thrust;

extern "C"
{

__global__ void CUDAlogkernel(const double a, const double b, const int nu, const double *u, double *x, double *y, double *ret)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int n = sizeof(x)/sizeof(x[0]);

    const double pi = M_PI;
    const double lengthd = abs(b-a);
    const double C = 0.5*lengthd;

    complex<double> *z,*yv,*yk,*ykp1;

    z = new complex<double>[n];
    yv = new complex<double>[n];
    yk = new complex<double>[n];
    ykp1 = new complex<double>[n];


    z[i] = complex<double>(x[i],y[i]);
    z[i] = (a + b - 2.0*z[i])/(a - b);  // tocanonical(u,z)

    if (z[i].real() <= 1.0 && z[i].real() >= -1.0 && abs(z[i].imag()) <= 2.0e-14) {
        yv[i] = z[i]+complex<double>(0.0,1.0)*sqrt(1.0-z[i])*sqrt(z[i]+1.0);
    }
    else {
        yv[i] = z[i] - sqrt(z[i]-1.0)*sqrt(z[i]+1.0);  // updownjoukowskyinverse(true,z)
    }

    yk[i] = yv[i];
    ykp1[i] = yk[i]*yk[i];


    if ( nu >= 0 ) {
        ret[i] = -u[0]*log(abs(2.0*yk[i]/C));  // -logabs(2y/C)
        if ( nu >= 1 ) {
            ret[i] += -u[1]*yk[i].real();  // -real(yk)
            if ( nu >= 2 ) {
                ret[i] += u[2]*(log(abs(2.0*yk[i]/C))-0.5*ykp1[i].real()); // -ret[1]-.5real(ykp1)
                if ( nu >= 3) {
                    for (int nun = 3; nun<nu; nun++) {
                        ykp1[i] *= yv[i];
                        ret[i] += u[nun]*( yk[i].real()/(nun-2.0)-ykp1[i].real()/(nun-0.0) ); // real(yk)/(n-3)-real(ykp1)/(n-1)
                        yk[i] *= yv[i];
                    }
                }
            }
        }
    }
    ret[i] *= pi*C;
}

} // extern "C"
